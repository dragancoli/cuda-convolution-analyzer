#include <iostream>
#include <fstream>
#include <vector>
#include <cstdint>
#include <chrono>
#include <algorithm>  
#include <hip/hip_runtime.h>

// Strukture za BMP
#pragma pack(push, 1)
struct BMPHeader {
    uint16_t fileType{0x4D42}; 
    uint32_t fileSize{0};
    uint16_t reserved1{0};
    uint16_t reserved2{0};
    uint32_t offsetData{0};
};

struct BMPInfoHeader {
    uint32_t size{0};
    int32_t width{0};
    int32_t height{0};
    uint16_t planes{1};
    uint16_t bitCount{0};
    uint32_t compression{0};
    uint32_t sizeImage{0};
    int32_t xPixelsPerMeter{0};
    int32_t yPixelsPerMeter{0};
    uint32_t colorsUsed{0};
    uint32_t colorsImportant{0};
};
#pragma pack(pop)

// Struktura za BMP podatke
struct BMP {
    BMPHeader header;
    BMPInfoHeader infoHeader;
    std::vector<uint8_t> data;

    bool load(const std::string &filename) {
        std::ifstream file(filename, std::ios::binary);
        if (!file) {
            std::cerr << "Error opening file: " << filename << std::endl;
            return false;
        }
        file.read(reinterpret_cast<char*>(&header), sizeof(header));
        file.read(reinterpret_cast<char*>(&infoHeader), sizeof(infoHeader));

        if (header.fileType != 0x4D42) {
            std::cerr << "Error: Not a BMP file" << std::endl;
            return false;
        }

        data.resize(infoHeader.width * infoHeader.height * 3);
        file.seekg(header.offsetData, file.beg);
        file.read(reinterpret_cast<char*>(data.data()), data.size());
        return true;
    }

    bool save(const std::string &filename) {
        std::ofstream file(filename, std::ios::binary);
        if (!file) {
            std::cerr << "Error opening file: " << filename << std::endl;
            return false;
        }
        file.write(reinterpret_cast<const char*>(&header), sizeof(header));
        file.write(reinterpret_cast<const char*>(&infoHeader), sizeof(infoHeader));
        file.write(reinterpret_cast<const char*>(data.data()), data.size());
        return true;
    }
};


// Pomoćna funkcija za clamp
__device__ int clampInt(int value, int low, int high) {
    if (value < low)   return low;
    if (value > high)  return high;
    return value;
}

// CUDA kernel za konvoluciju
__global__ void gpuConvolutionKernel(
    const uint8_t* input, 
    uint8_t* output, 
    const int* kernel,
    int width, 
    int height,
    int kernelSize
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int kHalf = kernelSize / 2;

    if (x < kHalf || x >= (width - kHalf) || y < kHalf || y >= (height - kHalf)) {
        return;
    }

    int sumR = 0, sumG = 0, sumB = 0;

    for (int ky = -kHalf; ky <= kHalf; ky++) {
        for (int kx = -kHalf; kx <= kHalf; kx++) {
            int inX = x + kx;
            int inY = y + ky;
            int pixelIndex = (inY * width + inX) * 3;
            int kernelValue = kernel[(ky + kHalf) * kernelSize + (kx + kHalf)];

            sumR += static_cast<int>(input[pixelIndex])     * kernelValue;
            sumG += static_cast<int>(input[pixelIndex + 1]) * kernelValue;
            sumB += static_cast<int>(input[pixelIndex + 2]) * kernelValue;
        }
    }

    sumR = clampInt(sumR, 0, 255);
    sumG = clampInt(sumG, 0, 255);
    sumB = clampInt(sumB, 0, 255);

    int outIdx = (y * width + x) * 3;
    output[outIdx]     = static_cast<uint8_t>(sumR);
    output[outIdx + 1] = static_cast<uint8_t>(sumG);
    output[outIdx + 2] = static_cast<uint8_t>(sumB);
}

// GPU Funkcija za konvoluciju
void applyConvolutionGPU(
    const std::vector<uint8_t>& input,
    std::vector<uint8_t>& output,
    const std::vector<int>& kernel,
    int width,
    int height,
    int kernelSize
) {
    uint8_t *d_input = nullptr, *d_output = nullptr;
    int* d_kernel = nullptr;

    size_t imageSize = static_cast<size_t>(width) * height * 3 * sizeof(uint8_t);
    size_t kernelBytes = kernel.size() * sizeof(int);

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);
    hipMalloc(&d_kernel, kernelBytes);

    hipMemcpy(d_input,  input.data(),  imageSize,    hipMemcpyHostToDevice);
    hipMemcpy(d_output, output.data(), imageSize,    hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, kernel.data(), kernelBytes,  hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((width  + block.x - 1) / block.x,
              (height + block.y - 1) / block.y);

    gpuConvolutionKernel<<<grid, block>>>(d_input, d_output, d_kernel, width, height, kernelSize);
    hipDeviceSynchronize();

    hipMemcpy(output.data(), d_output, imageSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);
}

int main(int argc, char *argv[]) {
    std::string inputFile  = "test.bmp";
    std::string outputFile = "output.bmp";

    std::vector<int> kernel = {
        -1, -1,  -1,
        -1,  8, -1,
        -1, -1, -1
    };

    if (argc == 1) {
        // Nema argumenata, koristi podrazumevane vrednosti
        std::cout << "Using default input: test.bmp and output: output.bmp\n";
    }
    else if (argc >= 3) {
        inputFile  = argv[1];
        outputFile = argv[2];
        if (argc > 3) {
            kernel.clear();
            for (int i = 3; i < argc; ++i) {
                kernel.push_back(std::stoi(argv[i]));
            }
        }
    } else {
        std::cerr << "Usage: " << argv[0]
                  << " <input.bmp> <output.bmp> [kernel values...]\n";
        return 1;
    }

    BMP bmp;
    if (!bmp.load(inputFile)) {
        std::cerr << "Failed to load: " << inputFile << std::endl;
        return 1;
    }

    std::vector<uint8_t> output(bmp.data.size(), 0);

    for (int i = 0; i < 10; i++) {
        applyConvolutionGPU(bmp.data, output, kernel, bmp.infoHeader.width, bmp.infoHeader.height, 3);
    }

    double times[10];
    for (int i = 0; i < 10; i++) {
        auto start = std::chrono::high_resolution_clock::now();
        applyConvolutionGPU(bmp.data, output, kernel, bmp.infoHeader.width, bmp.infoHeader.height, 3);
        auto end = std::chrono::high_resolution_clock::now();
        times[i] = std::chrono::duration<double>(end - start).count();
    }

    double sum = 0;
    for (int i = 0; i < 10; i++) {
        sum += times[i];
    }
    
    double avg_time = sum/10;

    double variance = 0;
    for (int i = 0; i < 10; i++) {
        variance += pow(times[i] - avg_time, 2);
    }
    variance = variance/10;

    std::cout << "Vrijeme: " << avg_time  << " Varijansa: " << variance  << std::endl;


    bmp.data = output;
    if (!bmp.save(outputFile)) {
        std::cerr << "Failed to save: " << outputFile << std::endl;
        return 1;
    }
    return 0;
}